#include "lattice_net/HashTable.cuh"

#include <hip/hip_runtime.h>
// #include "torch/torch.h" 

//my stuff 
#include "lattice_net/kernels/HashTableGPU.cuh"



HashTable::HashTable(const int capacity):
    m_capacity(capacity), 
    // m_pos_dim(-1),
    m_impl( new HashTableGPU() ),
    m_nr_filled_is_dirty(true),
    m_nr_filled(-1)
    {
}


void HashTable::init(int pos_dim, int val_dim){

    // CHECK()

    // m_capacity=capacity;
    // m_pos_dim=pos_dim;
    m_impl=std::make_shared<HashTableGPU>( m_capacity, pos_dim );

    // m_keys_tensor=register_buffer("keys", torch::zeros({capacity, pos_dim}).to(torch::kInt32) ); //TODO should it be short so kInt16 as in the original implementation
    // torch::zeros({m_capacity, pos_dim  }, torch::dtype(torch::kFloat32).device(torch::kCUDA, 0) )
    m_keys_tensor=register_buffer("keys",   torch::zeros({m_capacity, pos_dim  }, torch::dtype(torch::kInt32).device(torch::kCUDA, 0))    ); //TODO should it be short so kInt16 as in the original implementation
    m_values_tensor=register_buffer("values", torch::zeros({m_capacity, val_dim  }, torch::dtype(torch::kFloat32).device(torch::kCUDA, 0))   );
    m_entries_tensor=register_buffer("entries",   torch::zeros({m_capacity  }, torch::dtype(torch::kInt32).device(torch::kCUDA, 0))    );
    m_nr_filled_tensor=register_buffer("nr_filled", torch::zeros({1}, torch::dtype(torch::kInt32).device(torch::kCUDA, 0))  );
    m_nr_filled_is_dirty=true;

    // m_keys_tensor=m_keys_tensor.to("cuda");
    // m_values_tensor=m_values_tensor.to("cuda");
    // m_entries_tensor=m_entries_tensor.to("cuda");
    // m_nr_filled_tensor=m_nr_filled_tensor.to("cuda");


    clear();
    update_impl();


}

void HashTable::clear_only_values(){
    if(is_initialized()){
        m_values_tensor.fill_(0);
        //m_nr_filled_is_dirty=true;
    }
}

void HashTable::clear(){
    if(is_initialized()){
        m_values_tensor.fill_(0);
        m_keys_tensor.fill_(0);
        m_entries_tensor.fill_(-1);
        m_nr_filled_tensor.fill_(0);
        m_nr_filled_is_dirty=true;
    }
}

void HashTable::clear_only_values(){
    if(is_initialized()){
        m_values_tensor.fill_(0);
        // m_nr_filled_is_dirty=true;
    }
}

bool HashTable::is_initialized(){
    if(m_keys_tensor.defined() ){
        return true;
    }else{
        return false;
    }

}

void HashTable::update_impl(){
    m_impl->m_capacity = m_capacity;
    if(m_keys_tensor.defined()){
        m_impl->m_keys = m_keys_tensor.data_ptr<int>();
    }
    if(m_values_tensor.defined()){
        m_impl->m_values = m_values_tensor.data_ptr<float>();
    }
    if(m_entries_tensor.defined()){
        m_impl->m_entries = m_entries_tensor.data_ptr<int>();
    }
    if(m_nr_filled_tensor.defined()){
        m_impl->m_nr_filled = m_nr_filled_tensor.data_ptr<int>();
    }

    CHECK( m_keys_tensor.defined() )<<" We need the keys tensor to be defined here. Please use hash_table.init() first.";

    m_impl->m_pos_dim = m_keys_tensor.size(1);

}



//getters 
int HashTable::pos_dim(){
    return m_keys_tensor.size(1);
}
int HashTable::val_dim(){
    return m_values_tensor.size(1);
}
int HashTable::capacity(){
    return m_keys_tensor.size(0);
}



//setters
void HashTable::set_values(const torch::Tensor& new_values){
    m_values_tensor=new_values.contiguous();
    update_impl();
}
 